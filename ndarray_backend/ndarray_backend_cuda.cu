#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>
#include <cmath>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);
typedef ssize_t ptrdiff_t;

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides

// given specification of an array in a matrix, and the index of the will be compact array,
// return the index in the matrix that corresponds to this compact array index.
__device__ size_t compact_to_non_compact_index(size_t compact_index, CudaVec shape, CudaVec strides, size_t offset) {
  // can't use vector here because we are in CUDA?!
  // std::vector<uint32_t> matrix_indices; // compact_index = 12, shape = (3, 2, 5) -> matrix_indices = (1, 0, 2)
  CudaVec compact_strides;
  size_t num_dimensions = shape.size;
  compact_strides.size = num_dimensions;
  size_t dimension_prod = 1;
  for (int i = num_dimensions - 1; i >= 0; i--) {
    compact_strides.data[i] = dimension_prod;
    dimension_prod *= shape.data[i];
  }
  size_t result_index = offset;
  size_t remainder = compact_index;
  size_t dim_i_index;
  for (size_t i = 0; i < shape.size; i++) {
    dim_i_index = remainder / compact_strides.data[i]; // the index in dimension i in matrix
    remainder %= compact_strides.data[i];
    result_index += dim_i_index * strides.data[i];
  }
  return result_index;
}


__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  ssize_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN YOUR SOLUTION
  if (gid < size) {
    size_t a_index = compact_to_non_compact_index(gid, shape, strides, offset);
    out[gid] = a[a_index];
  }
  /// END YOUR SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}

__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size) {
    size_t out_index = compact_to_non_compact_index(gid, shape, strides, offset);
    out[out_index] = a[gid];
  }
}


void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(a.size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape),
                                              VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}


__global__ void ScalarSetitemKernel(scalar_t val, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size) {
    size_t out_index = compact_to_non_compact_index(gid, shape, strides, offset);
    out[out_index] = val;
  }
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape),
                                               VecToCuda(strides), offset);
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

/// BEGIN YOUR SOLUTION
__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * b[gid];
}

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * val;
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / b[gid];
}

void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / val;
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = pow(a[gid], val);
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  // if (gid < size) out[gid] = std::max(a[gid], b[gid]); -- max not available in cuda kernel
  if (gid < size) out[gid] = a[gid] > b[gid] ? a[gid] : b[gid];
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  // if (gid < size) out[gid] = std::max(a[gid], val);
  if (gid < size) out[gid] = a[gid] > val ? a[gid] : val;
}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseEqKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] == b[gid];
}

void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseEqKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarEqKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] == val;
}

void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarEqKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseGeKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] >= b[gid];
}

void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseGeKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarGeKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] >= val;
}

void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarGeKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseLogKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = log(a[gid]);
}

void EwiseLog(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseLogKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseExpKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = exp(a[gid]);
}

void EwiseExp(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseExpKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseTanhKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = tanh(a[gid]);
}

void EwiseTanh(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  EwiseTanhKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}
/// END YOUR SOLUTION

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

// implemnets shared memory + cooperative fetching
// each thread handles computation of a single element in C,
// and each thread block handles TILE x TILE computation in C.
// It is possible to have each thread handling computation of V x V elements in C,
// but we are trying to keep thing simple here.
__global__ void MatmulKernelSharedMemory(const scalar_t* A, const scalar_t* B, scalar_t* C, uint32_t M, uint32_t N, uint32_t P) {
  // memory each thread block shares
  __shared__ scalar_t A_shared[TILE][TILE];
  __shared__ scalar_t B_shared[TILE][TILE];

  // the number of elements to advance in each dimension due to the block idx we are on.
  size_t blockIdxRow = blockIdx.x * TILE;
  size_t blockIdxCol = blockIdx.y * TILE;

  // the number of elements to advance in each dimension due to the thread idx we are on.
  size_t threadIdxRow = threadIdx.x;
  size_t threadIdxCol = threadIdx.y;

  // register tiling: loading a vector at a time to GPU registers
  scalar_t a[TILE];
  scalar_t b[TILE];

  // the result of computation this thread is responsible for. which is DotProduct(A[blockIdxRow + threadIdxRow, :], B[:, blockIdxCol + threadIdxCol])
  scalar_t result = 0;

  // k is the index we are summing over. This k is for the block summation
  for (size_t k = 0; k < N; k += TILE) {
    __syncthreads();
    // cooperative fetching here.
    // thread block is TILE x TILE, and A_shared is TILE x TILE. each thread only needs
    // to fetch one element

    // A_shared[:][:] = A[blockIdxRow : blockIdxRow + TILE][k: k + TILE];
    if (blockIdxRow + threadIdxRow < M && k + threadIdxCol < N) {
      // A_shared[threadIdxRow][threadIdxCol] = A[blockIdxRow + threadIdxRow][k + threadIdxCol];
      A_shared[threadIdxRow][threadIdxCol] = A[(blockIdxRow + threadIdxRow) * N + k + threadIdxCol];
    } else {
      // elegant way to deal with out of index. If out of index
      // set the corresponding entry to 0, which won't get added to result
      // when multiplied.
      A_shared[threadIdxRow][threadIdxCol] = 0;
    }

    // B_shared[:][:] = B[k: k + TILE][blockIdxCol : blockIdxCol + TILE];
    if (k + threadIdxRow < N && blockIdxCol + threadIdxCol < P) {
      // B_shared[threadIdxRow][threadIdxCol] = B[k + threadIdxRow][blockIdxCol + threadIdxCol]
      B_shared[threadIdxRow][threadIdxCol] = B[(k + threadIdxRow) * P + blockIdxCol + threadIdxCol];
    } else {
      B_shared[threadIdxRow][threadIdxCol] = 0;
    }

    __syncthreads();

    // non register tiling:
    // i = summation index for the thread
    // for (size_t i = 0; i < TILE; i++) {
    //   result += A_shared[threadIdxRow][i] * B_shared[i][threadIdxCol];
    // }

    for (size_t i = 0; i < TILE; i++) {
      a[i] = A_shared[threadIdxRow][i];
      b[i] = B_shared[i][threadIdxCol];
    }
    for (size_t i = 0; i < TILE; i++) {
      result += a[i] * b[i];
    }
  }

  
  if (blockIdxRow + threadIdxRow < M && blockIdxCol + threadIdxCol < P) {
    // C[blockIdxRow + threadIdxRow, blockIdxCol + threadIdxCol] = result;
    C[(blockIdxRow + threadIdxRow) * P + blockIdxCol + threadIdxCol] = result;
  }
}

// simply parallelize the computation of each element in matrix C to each thread.
__global__ void MatmulKernelSimple(const scalar_t* A, const scalar_t* B, scalar_t* C, uint32_t M, uint32_t N, uint32_t P) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t row = gid / P;
  size_t col = gid % P;
  if (row < M && col < P) {
    scalar_t out_value_row_col = 0;
    for (size_t k = 0; k < N; k++) {
      out_value_row_col += A[row * N + k] * B[k * P + col];
    }
    C[gid] = out_value_row_col;
  }
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN YOUR SOLUTION
  
  // shared memory:
  // each thread handles 1 element, each thread block handles TILE x TILE elements in result matrix
  dim3 blockDims(TILE, TILE);
  dim3 gridDims((M + TILE - 1) / TILE, (P + TILE - 1) / TILE); // how many blocks = how many elements / # elements in a block
  MatmulKernelSharedMemory<<<gridDims, blockDims>>>(a.ptr, b.ptr, out->ptr, M, N, P);

  // simple:
  // CudaDims dim = CudaOneDim(out->size);
  // MatmulKernelSimple<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P);

  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t out_size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < out_size) {
    size_t start_index = gid * reduce_size;
    size_t end_index = (gid + 1) * reduce_size;
    scalar_t reduce_result = a[start_index];
    for (size_t i = start_index; i < end_index; i++) {
      reduce_result = reduce_result > a[i] ? reduce_result : a[i];
    }
    out[gid] = reduce_result;
  }
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
  /// END YOUR SOLUTION
}

__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t out_size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < out_size) {
    size_t start_index = gid * reduce_size;
    size_t end_index = (gid + 1) * reduce_size;
    scalar_t reduce_result = 0;
    for (size_t i = start_index; i < end_index; i++) {
      reduce_result += a[i];
    }
    out[gid] = reduce_result;
  }
}


void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
  /// END YOUR SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}